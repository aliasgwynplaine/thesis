#include "hip/hip_runtime.h"
#include "cuda_test.cuh"


__global__ void hello_test() {
    int bid = blockIdx.x;
    int tid = threadIdx.x;

    printf("[[%d], %d]: Hello, friend, from GPU!\n", bid, tid);
}

extern "C" {
    int hello_test_wrapper() {
        debug("Calling kernel hello_test from the hello_test_wrapper fux...");
        hello_test<<<2,1>>>();
        hipDeviceSynchronize();
        return 0;
    }
}
