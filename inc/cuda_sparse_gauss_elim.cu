#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "matrix.h"
#include "pol.h"
#include "cuda_outils.h"

__global__ void sparse_trsm(stbm_t * a, srbm_t * b) {

}


__global__ void sparse_gauss_elim(hrbm_t * d) {

}


void cuda_csr_rref(csr_t * m, dctx_t * ctx) {
    dctx_t * ctx = csr_analyse(m);
    idx_t cc = 0;

    for (idx_t k = 0; cc < ctx->npiv; k++) {
        if (ctx->rpc[k] != -1 && ctx->rpc[k] != k) {
            printf("swaping %d <> %d -> ", k, ctx->rpc[k]);
            csr_swap_col(m, ctx->rpc[k], k);
        }
        cc++;
    }

    /* copy mem from host to device */
    hipError_t err = hipSuccess;

    stm_t * b = NULL;
    CUDAMALLOC(b, sizeof(stm_t), err);

    


    /* call fux to operate mat on device */

    /* copy mem from dev to host */


    /* free mem */

    dctx_free(ctx);
    CUDAFREE(b, err);
}


void cuda_sparse_trsm(stbm_t * a, srbm_t * b) {
    /*
        prepare blocks and threads
        call the fux...
    */

}

void cuda_sparse_gauss_elim(hrbm_t * d) {
    /*
        prepare blocks and threads
        call the fux...
    */

}
